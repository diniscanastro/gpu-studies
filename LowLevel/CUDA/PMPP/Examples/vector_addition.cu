
#include <hip/hip_runtime.h>
#include <stdio.h>

// Compute vector sum h_C = h_A+h_B
__global__
void vecAdd(float* A, float* B, float* C, int n){
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    if (i<n){
        C[i] = A[i] + B[i];
    } 
}

int main(){
    // Memory allocation for h_A, h_B, and h_C
    // Allocate memory
    const int N = 1000;
    const int N_size = sizeof(float) * N;
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(N_size);
    h_B = (float*)malloc(N_size);
    h_C = (float*)malloc(N_size);

    // Initialize input arrays
    for (int i = 0; i < N; i++){
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N_size);
    hipMalloc((void**)&d_B, N_size);
    hipMalloc((void**)&d_C, N_size);

    // Copy input arrays to device memory
    hipMemcpy(d_A, h_A, N_size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N_size, hipMemcpyHostToDevice);

    // Run kernel
    vecAdd<<<ceil(N/256.0),256>>>(d_A, d_B, d_C, N);

    // Copy output array to host memory
    hipMemcpy(h_C, d_C, N_size, hipMemcpyDeviceToHost);

    // Print result
    for (int i=0; i < N; i++){
        printf("%f\n", h_C[i]);
    }

    // Free device allocated memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host allocated memory
    free(h_A);
    free(h_B);
    free(h_C);
}