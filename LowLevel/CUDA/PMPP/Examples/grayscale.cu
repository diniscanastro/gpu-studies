#include "hip/hip_runtime.h"
#include <stdio.h>
#include "include/auxiliary.c"

#define CHANNELS 3

// Device Kernel
__global__
void colorToGreyscaleConversion(unsigned char *Pout, unsigned char *Pin, int height, int width){
    int col = blockDim.x*blockIdx.x + threadIdx.x;
    int row = blockDim.y*blockIdx.y + threadIdx.y;

    if (col < width && row < height){
        int offset = (row * width + col) * CHANNELS;

        unsigned char r = Pin[offset];
        unsigned char g = Pin[offset + 1];
        unsigned char b = Pin[offset + 2];

        unsigned char grey_value = 0.21f*r + 0.71f*g + 0.07f*b;

        Pout[offset] = grey_value;
        Pout[offset + 1] = grey_value;
        Pout[offset + 2] = grey_value;
    } 
}


int main(){    
    // Initialize host memory input values from file
    const char *input_file_path = "test_images/test_image.ppm";
    struct PPMImage input_image = read_ppm(input_file_path);

    // Allocate device memory
    int total_size = input_image.height * input_image.width * CHANNELS;
    unsigned char *Pin, *Pout;
    hipMalloc((void**)&Pin, total_size);
    hipMalloc((void**)&Pout, total_size);

    // Transfer input to device memory
    hipMemcpy(Pin, input_image.content, total_size, hipMemcpyHostToDevice);

    // Run Kernel
    dim3 dimGrid(ceil(input_image.height/16.0), ceil(input_image.width/16.0), 1);
    dim3 dimBlock(16, 16, 1);
    colorToGreyscaleConversion<<<dimGrid,dimBlock>>>(Pout, Pin, input_image.height, input_image.width);

    // Transfer output to host memory
    struct PPMImage output_image = copy_ppm(input_image);
    hipMemcpy(output_image.content, Pout, total_size, hipMemcpyDeviceToHost);
    
    // Write results to new file
    const char* output_file_path = "outputs/output_test.ppm";
    write_ppm(output_file_path, output_image);

    // De-allocate device memory
    hipFree(Pin);
    hipFree(Pout);

    // De-allocate host memory
    free(input_image.content);
    free(output_image.content);
}